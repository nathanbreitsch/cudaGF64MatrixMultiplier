
#include <hip/hip_runtime.h>
#include <stdio.h>

//define matrix type
typedef struct{
  int row_count;
  int column_count;
  int* elements;
} Matrix;


__global__ void multiply_kernel_stupid(const Matrix left, const Matrix right, Matrix result);
Matrix ones(int row_count, int column_count);
Matrix multiply(Matrix left, Matrix right);
void print_matrix(Matrix mat);

int main(){
  //make the matrices you want to multiply
  Matrix A = ones(10, 5);
  Matrix B = ones(5, 10);
  Matrix result = multiply(A, B);
  print_matrix(A);
  print_matrix(B);
  print_matrix(result);
}

//stupid kernel, one thread per result cell, global memory, no use of spacial locality.
__global__ void multiply_kernel_stupid(const Matrix left, const Matrix right, Matrix result){
  int sum = 0;
  int row_index = blockIdx.y * blockDim.y + threadIdx.y;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = 0; i < left.column_count; i++){
    int left_index = row_index * left.column_count  + i;
    int right_index = column_index + i * right.column_count;
    sum += left.elements[left_index] * right.elements[right_index];
  }
  result.elements[row_index * result.row_count + column_index] = sum;

}

Matrix multiply(Matrix left, Matrix right){
  hipError_t error;
  //step 1: allocate memory on the kernel for left, right, result
  Matrix left_d, right_d;
  left_d.row_count = left.row_count;
  left_d.column_count = left.column_count;
  size_t left_size = left.row_count * left.column_count * sizeof(int);
  error = hipMalloc((void**) &left_d.elements, left_size);
  if(error != hipSuccess){
    printf("error allocating left matrix\n");
    printf("CUDA error: %s\n", hipGetErrorString(error));
  }


  right_d.row_count = right.row_count;
  right_d.column_count = right.column_count;
  size_t right_size = right.row_count * right.column_count * sizeof(int);
  error = hipMalloc((void**) &right_d.elements, right_size);
  if(error != hipSuccess){ printf("error allocating right matrix\n"); }

  //step 2: allocate memory on the host and device for result
  Matrix result, result_d;
  result.row_count = result_d.row_count = left.row_count;
  result.column_count = result_d.column_count = right.column_count;
  size_t result_size = result.row_count * result.column_count * sizeof(int);
  result.elements = (int*) malloc(result_size);
  error = hipMalloc((void**) &result_d.elements, result_size);
  if(error != hipSuccess){ printf("error allocating matrix\n"); }

  //step 3: copy left and right to device
  error = hipMemcpy(left_d.elements, left.elements, left_size, hipMemcpyHostToDevice);
  if(error != hipSuccess){ printf("error copying left matrix\n"); }
  error = hipMemcpy(right_d.elements, right.elements, right_size, hipMemcpyHostToDevice);
  if(error != hipSuccess){ printf("error copying right matrix\n"); }

  //step 4: launch kernel



  dim3 block_dims(32, 32);
  dim3 grid_dims(result.column_count / block_dims.x + 1, result.row_count / block_dims.y + 1);
  multiply_kernel_stupid <<<grid_dims, block_dims>>> (left_d, right_d, result_d);

  //step 5: copy results back to host
  error = hipMemcpy(result.elements, result_d.elements, result_size, hipMemcpyDeviceToHost);
  if(error != hipSuccess){
  	printf("error copying result matrix\n");
  	printf("CUDA error: %s\n", hipGetErrorString(error));
  }
  return result;
}

Matrix ones (int row_count, int column_count){
  Matrix result;
  result.row_count = row_count;
  result.column_count = column_count;
  result.elements = (int*) malloc(row_count * column_count * sizeof(int));
  for(int i = 0; i < row_count * column_count; i++){
    result.elements[i] = 1;
  }
  return result;
}

void print_matrix(Matrix mat){
  int num_elements = mat.row_count * mat.column_count;
  for(int i = 0; i < num_elements; i++){
    printf(" %d", mat.elements[i]);
    if(!((i + 1) % mat.column_count)){ printf("\n"); }
  }
}
